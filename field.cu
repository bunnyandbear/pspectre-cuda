#include "hip/hip_runtime.h"
/*
 * SpectRE - A Spectral Code for Reheating
 * Copyright (C) 2009-2010 Hal Finkel, Nathaniel Roth and Richard Easther
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED ``AS IS'' AND ANY EXPRESS OR IMPLIED WARRANTIES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
 * AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL
 * THE AUTHORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
 * ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "pow.hpp"
#include "field.hpp"

#include <cstdlib>
#include <cstring>
#include <cmath>

#include <iostream>
#include <thrust/device_free.h>
#include <thrust/fill.h>

using namespace std;

void print_memory_usage();

template <typename R>
void field<R>::construct(field_size &fs_)
{
	fs = fs_;
	ldl = 2*(fs.n/2+1);

#ifdef DEBUG
	cout << "\nConstructing field " << (name ? name : "unknown") << endl;
	cout << "Number of grid points: " << fs.n << endl;
	cout << "Memory usage before hipMalloc:" << endl;
	print_memory_usage();
#endif
	hipError_t ret = hipMalloc(&raw_ptr,
				     fs.total_momentum_gridpoints * sizeof(fftw_complex));
	if (ret != hipSuccess) {
		cout << "hipMalloc() failed. GPUassert: "
		     << hipGetErrorString(ret) << endl;
	}
#ifdef DEBUG
	cout << "Memory usage after hipMalloc:" << endl;
	print_memory_usage();
#endif
	mdata = gpu_array_accessor_fftw_complex((fftw_complex *) raw_ptr);
	data = gpu_array_accessor_double(raw_ptr);
	fill0();

	m2p_plan.construct(fs.n, fs.n, fs.n, mdata.ptr, data.ptr, false);
	p2m_plan.construct(fs.n, fs.n, fs.n, data.ptr, mdata.ptr, false);
}

template <typename R>
field<R>::~field()
{
#ifdef DEBUG
	cout << "Destructing field " << (name ? name : "unknown") << endl;
	cout << "Memory usage before hipFree:" << endl;
	print_memory_usage();
#endif
	hipError_t ret = hipFree(&raw_ptr);
	if (ret != hipSuccess) {
		cout << "hipFree() failed. GPUassert: "
		     << hipGetErrorString(ret) << endl;
	}
#ifdef DEBUG
	cout << "Memory usage after hipFree:" << endl;
	print_memory_usage();
#endif
}

/* (x*y) * (z)
 * (n^2) * (n/2+1)
 * BLK     THR
 *         NO-PADDING
 */
__global__ void momentum_divby_kernel(fftw_complex *mdata, double v)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	mdata[idx][0] /= v;
	mdata[idx][1] /= v;
}

/* (x*y) * (z)
 * (n^2) * (2*(n/2+1))
 * BLK     THR
 *         PADDED from n to (2*(n/2+1))
 */
__global__ void position_divby_kernel(double *data, double v, int ldl)
{
	int idx = ldl * blockIdx.x + threadIdx.x;
	data[idx] /= v;
}

template <typename R>
void field<R>::divby(R v)
{
	int n = fs.n;
	if (state == momentum) {
		momentum_divby_kernel<<<n*n, n/2+1>>>(mdata.ptr, v);
	} else if (state == position) {
		position_divby_kernel<<<n*n, n>>>(data.ptr, v, ldl);
	}
}

template <typename R>
void field<R>::switch_state(field_state state_)
{
	if (state_ == uninitialized) {
		state = uninitialized;
	} else if (state == uninitialized) {
		state = state_;
	} else if ((state == position) && (state_ == momentum)) {
		state = momentum;
		p2m_plan.execute();
	} else if ((state == momentum) && (state_ == position)) {
		state = position;
		m2p_plan.execute();
		divby(fs.total_gridpoints);
	}
}

template <typename R>
void field<R>::fill0()
{
	hipError_t ret = hipMemset(raw_ptr, 0,
				     fs.total_momentum_gridpoints * sizeof(fftw_complex));
	if (ret != hipSuccess) {
		cout << "fill0: hipMemset() failed. GPUassert: "
		     << hipGetErrorString(ret) << endl;
	}
}

// Explicit instantiations
template class field<double>;
