#include "hip/hip_runtime.h"
/*
 * SpectRE - A Spectral Code for Reheating
 * Copyright (C) 2009-2010 Hal Finkel, Nathaniel Roth and Richard Easther
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED ``AS IS'' AND ANY EXPRESS OR IMPLIED WARRANTIES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
 * AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL
 * THE AUTHORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
 * ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "pow.hpp"
#include "field.hpp"

#include <cstdlib>
#include <cstring>
#include <cmath>

#include <iostream>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/fill.h>

using namespace std;

template <typename R>
void field<R>::construct(field_size &fs_)
{
	fs = fs_;
	ldl = 2*(fs.n/2+1);

	dev_ptr = thrust::device_malloc<fftw_complex>(fs.total_momentum_gridpoints);
	fftw_complex *raw_ptr = thrust::raw_pointer_cast(dev_ptr);
	mdata = gpu_array_accessor_fftw_complex(raw_ptr);
	data = gpu_array_accessor_double((double *) raw_ptr);
	fill0();

	m2p_plan.construct(fs.n, fs.n, fs.n, mdata.ptr, data.ptr, false);
	p2m_plan.construct(fs.n, fs.n, fs.n, data.ptr, mdata.ptr, false);
}

template <typename R>
field<R>::~field()
{
	thrust::device_free(dev_ptr);
}

/* (x*y) * (z)
 * (n^2) * (n/2+1)
 * BLK     THR
 *         NO-PADDING
 */
__global__ void momentum_divby_kernel(fftw_complex *mdata, double v)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	mdata[idx][0] /= v;
	mdata[idx][1] /= v;
}

/* (x*y) * (z)
 * (n^2) * (2*(n/2+1))
 * BLK     THR
 *         PADDED from n to (2*(n/2+1))
 */
__global__ void position_divby_kernel(double *data, double v, int ldl)
{
	int idx = ldl * blockIdx.x + threadIdx.x;
	data[idx] /= v;
}

template <typename R>
void field<R>::divby(R v)
{
	int n = fs.n;
	if (state == momentum) {
		momentum_divby_kernel<<<n*n, n/2+1>>>(mdata.ptr, v);
	} else if (state == position) {
		position_divby_kernel<<<n*n, n>>>(data.ptr, v, ldl);
	}
}

template <typename R>
void field<R>::switch_state(field_state state_)
{
	if (state_ == uninitialized) {
		state = uninitialized;
	} else if (state == uninitialized) {
		state = state_;
	} else if ((state == position) && (state_ == momentum)) {
		state = momentum;
		p2m_plan.execute();
	} else if ((state == momentum) && (state_ == position)) {
		state = position;
		m2p_plan.execute();
		divby(fs.total_gridpoints);
	}
}

template <typename R>
void field<R>::fill0()
{
	auto ptr = thrust::device_ptr<double>(data.ptr);
	thrust::fill(ptr, ptr + 2*fs.total_momentum_gridpoints, 0.0);
}

// Explicit instantiations
template class field<double>;
