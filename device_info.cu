#include <stdio.h>

void print_cuda_info()
{
	int nr_dev = 0;

	hipGetDeviceCount(&nr_dev);
	if (nr_dev <= 0) {
		printf("==========================\n");
		printf("WARNING! WARNING! WARNING!\n");
		printf("No CUDA device found.\n");
		printf("==========================\n");
	}
	for (int i = 0; i < nr_dev; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
		       2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) /
		       1.0e6);
	}
}
