#include "hip/hip_runtime.h"
/*
 * SpectRE - A Spectral Code for Reheating
 * Copyright (C) 2009-2010 Hal Finkel, Nathaniel Roth and Richard Easther
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED ``AS IS'' AND ANY EXPRESS OR IMPLIED WARRANTIES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
 * AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL
 * THE AUTHORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
 * ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "pow.hpp"
#include "integrator.hpp"
#include "reduction_helper.hpp"

#include <hipfft/hipfftw.h>

using namespace std;

__global__ void integrator_kernel(fftw_complex *phi, fftw_complex *chi,
				  double *total_gradient_phi, double *total_gradient_chi,
				  int n, double dp)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int z = threadIdx.x;
	int px = x <= n/2 ? x : x - n;
	int py = y <= n/2 ? y : y - n;
	int pz = z;
	int idx = z + (n/2+1)*(y + n*x);

	double mom2 = pow2(dp)*(pow2(px) + pow2(py) + pow2(pz));
	mom2 *= (z == 0 || z == n/2) ? 1 : 2;

	total_gradient_phi[idx] += mom2*(pow2(phi[idx][0]) + pow2(phi[idx][1]));
	total_gradient_chi[idx] += mom2*(pow2(chi[idx][0]) + pow2(chi[idx][1]));
}

template <typename R>
void integrator<R>::avg_gradients(field<R> &phi, field<R> &chi,
				  R &avg_gradient_phi, R &avg_gradient_chi)
{
	phi.switch_state(momentum);
	chi.switch_state(momentum);

	auto total_gradient_phi_arr = double_array_gpu(NGRIDSIZE, NGRIDSIZE, NGRIDSIZE/2+1);
	auto total_gradient_chi_arr = double_array_gpu(NGRIDSIZE, NGRIDSIZE, NGRIDSIZE/2+1);
	dim3 num_blocks(NGRIDSIZE, NGRIDSIZE);
	dim3 num_threads(NGRIDSIZE/2+1, 1);
	integrator_kernel<<<num_blocks, num_threads>>>(phi.mdata.ptr, chi.mdata.ptr,
						       total_gradient_phi_arr.ptr(),
						       total_gradient_chi_arr.ptr(),
						       NGRIDSIZE, MP_DP);

	R total_gradient_phi = total_gradient_phi_arr.sum();
	R total_gradient_chi = total_gradient_chi_arr.sum();

	// Divide by total_gridpoints again to get *average* squared gradient and *average* potential energy.
	avg_gradient_phi = total_gradient_phi/pow<2, R>(NTOTAL_GRIDPOINTS);
	avg_gradient_chi = total_gradient_chi/pow<2, R>(NTOTAL_GRIDPOINTS);
}

// Explicit instantiations
template class integrator<double>;
